#include "hip/hip_runtime.h"
// Description
// TODO
// - define to typedef?
// - Timings
// - Correctness

#define precision_t float

__global__ 
void outer(precision_t* m, precision_t* n, precision_t* mn) {
}

int main(int argc, char* argv[]) {
  // Defaults
  // - k kernels
  int k = 1;
  // - m data on GPU
  size_t m = 1000;
  precision_t* d_m;
  // - n data on host
  size_t n = 1000;
  precision_t* h_n;
  precision_t* d_n;
  // - mn final matrix
  precision_t d_mn;
  precision_t h_mn;

  // Process Input
  if (argc > 1) {
    // TODO
  }
  // Allocate Memory
  // - Device
  hipMalloc(&d_m, m * sizeof(precision_t));
  // - Host
#if 0
  hipMallocManaged();
  malloc();
#else
  hipHostMalloc(&h_n, n * sizeof(precision_t));
  hipMalloc(&d_n, n * sizeof(precision_t));
#endif // 1
  // - Matrix
  hipHostMalloc(&h_mn, m * n * sizeof(precision_t));
  hipMalloc(&d_mn, m * n * sizeof(precision_t));

  // Generate Two Vectors
  // - Device
  // <<<kernel launch>>>
  // - Host
  for (int i = 0; i < n; i++) {
    //h_n[i] = rand;
  }

  // Copy Vector
#if 0
  hipMemcpyAsync();
#else
  hipMemcpy(d_n, h_n, n * sizeof(precision_t), hipMemcpyHostToDevice);
#endif // 1

  // Execute
  //outer<<<>>>(d_m, d_n, d_mn);

  // Copy Back
#if 0
  hipMemcpyAsync();
#else
  hipMemcpy(h_mn, d_mn, m * n * sizeof(precision_t), hipMemcpyDeviceToHost);
#endif // 1

  // Do something with the data to prevent optimization
  // mutiply by scalalr and print? Do some norm/reduction?
}

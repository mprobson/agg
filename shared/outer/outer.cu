#include "hip/hip_runtime.h"
// Description
//  Outer product
// TODO
// - Kernel Sizes
// - Timings
// Future
// - Correctness
// - (CUDA) Error Checking
// - Checking and Generating Doubles
#include <cstdlib>
#include <ctime>
#include <hiprand/hiprand_kernel.h>

typedef float precision_t;

__global__
void init(unsigned int seed, hiprandState* state) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed, idx, 0, &state[id]);
}

__global__
void generate(precision_t* m, size_t m, int rMax, hiprandState* state) {
  int idx = blockId.x * blockDim.x + threadIdx.x;
  hiprandState localState = state[id];
  for (int i = idx; i < m ; i += gridDim.x) {
    m[i] = hiprand_uniform(&localState) * rMax;
  }
  state[id] = localState;
}

__global__
void outer(precision_t* m, precision_t* n, precision_t* mn) {
}

int main(int argc, char* argv[]) {
  // Defaults
  // - k kernels
  int k = 1;

  // - m data on GPU
  size_t m = 1000;

  // - n data on host
  size_t n = 1000;

  // - r max number
  int rMax = 100;

  unsigned int hSeed = std::time(NULL);
  unsigned int dSeed = hSeed;

  // Process Input
  if (argc > 1) {
    // TODO
  }

  // Allocate Memory
#if 0
  hipMallocManaged();
  malloc();
#else
  // - Device
  precision_t* d_m;
  hipMalloc(&d_m, m * sizeof(precision_t));

  // - Host
  precision_t* h_n;
  hipHostMalloc(&h_n, n * sizeof(precision_t));
  precision_t* d_n;
  hipMalloc(&d_n, n * sizeof(precision_t));

  // - Matrix
  precision_t h_mn;
  hipHostMalloc(&h_mn, m * n * sizeof(precision_t));
  precision_t d_mn;
  hipMalloc(&d_mn, m * n * sizeof(precision_t));

  hiprandState* d_states;
  hipMalloc(&d_states, * sizeof(hiprandState));
#endif // 1

  //init<<<>>>(dSeed, d_states);

  // Generate Two Vectors
  // - Device
  //generate<<<>>>(d_m, m, rMax, d_states);

  // - Host
  std::srand(hSeed);
  for (int i = 0; i < n; i++) {
    h_n[i] = std::rand() / ((RAND_MAX + 1u) / rMax);
  }

  // Ensure random number generation finishes
  hipDeviceSynchronize();

  // Copy Vector
#if 0
  hipMemcpyAsync();
#else
  hipMemcpy(d_n, h_n, n * sizeof(precision_t), hipMemcpyHostToDevice);
#endif // 1

  // Execute
  //outer<<<>>>(d_m, d_n, d_mn);

  // Copy Back
#if 0
  hipMemcpyAsync();
#else
  hipMemcpy(h_mn, d_mn, m * n * sizeof(precision_t), hipMemcpyDeviceToHost);
#endif // 1

  // Do something with the data to prevent optimization
  // mutiply by scalar and print? Do some norm/reduction?

  hipFree(d_m);
  hipFree(d_n);
  hipFree(d_mn);
  hipFree(d_states);
  // free/hipFree h_*
}
